//Header files
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<sys/time.h>

//Macros
#define ROWS 400	//No. of rows in orig image
#define COLS 640	//No. of cols in orig image
#define PLANES 3	//No. of planes in orig image
#define MASTER 0	//In Future, this code can be used with MPI

#define CUDA_SAFE_CALL(call)														 \
			do{                                                                      \
                 hipError_t err = call;                                             \
                 if(err != hipSuccess)                                              \
                 {                                                                   \
                        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                         __FILE__, __LINE__, hipGetErrorString( err) );             \
                         exit(1);                                                    \
                 }                                                                   \
               } while (0)   


/*This function converts pixel intensity from RGB value to Gray scale */
void RGB2GRAY(uchar4 *In, uint8_t *Gray){
	
	int ii=0;	
	for(ii=0; ii<ROWS*COLS; ii++){
		//Convert to Gray value
		Gray[ii] = (uint8_t)(0.3*In[ii].x + 0.59*In[ii].y + 0.11*In[ii].z);
		//printf("R = %d G = %d B = %d GR = %d \n", In[ii].x, In[ii].y, In[ii].z, Gray[ii]);
	}
}	//End of RGB2GRAY 


/*This function converts pixel intensity from RGB value to Gray scale */
__global__ void CUDA_RGB2GRAY(uchar4 *In, uint8_t *Gray){
	
	const long ii = threadIdx.x + blockDim.x*blockIdx.x;	//Thread initialization
	/*blockDim.x is no of threads per block */
	
	if(ii<ROWS*COLS){	//If thresads are too many
		//Convert to Gray value
		Gray[ii] = (uint8_t)(0.3*In[ii].x + 0.59*In[ii].y + 0.11*In[ii].z);
		//printf("R = %d G = %d B = %d GR = %d ii = %ld \n", In[ii].x, In[ii].y, In[ii].z, Gray[ii], ii);
	}
}	//End of CUDA_RGB2GRAY 


/*This function checks if the device (GPU) is available */
int CheckDevice(int rank)
{
        int DeviceCount, Device;
        struct hipDeviceProp_t Properties;

        hipGetDeviceCount(&DeviceCount);
        if(DeviceCount >= 1)
        {
                hipGetDevice(&Device);
                hipGetDeviceProperties(&Properties, Device);
                printf("Processor with rank %d has the Device by name %s and computation is done on this device \n",rank, Properties.name);
        }

        return(DeviceCount);
}//End of CheckDevice


int main(void){
	
	int ii=0, jj=0;	//Counter variables
	FILE *fptr;	//File pointer
	
	//Host variables
	uchar4 *InMat;
	uint8_t *OutMat;
	
	//Device variables
	uchar4 *d_InMat;
	uint8_t *d_OutMat;
	int DeviceStatus=0;
	int BLOCKSIZE, GRIDSIZE;
	
	//For timing
	struct timeval start, end;
	
	//Read the file
	if((fptr=fopen("./Input.txt","r")) == NULL){		
		printf("Input.txt file does not exist in the current folder. \n");
		return 0;
	}
	
	//Allocating memory
	InMat = (uchar4 *)malloc(ROWS*COLS*sizeof(uchar4));
	OutMat = (uint8_t *)malloc(ROWS*COLS*sizeof(uint8_t));
		
	printf("Reading matrix from Input.txt \n");
	for(ii=0; ii<ROWS; ii++){
		for(jj=0; jj<COLS; jj++){
						
			fscanf(fptr,"%d",&InMat[ii*COLS+jj].x);
			fscanf(fptr,"%d",&InMat[ii*COLS+jj].y);
			fscanf(fptr,"%d",&InMat[ii*COLS+jj].z);
			
		}		
	}
	printf("Input matrix is read! \n");
	fclose(fptr);	//Close the file
	
	//Check if device (GPU) is avialable
	DeviceStatus = CheckDevice(MASTER);
	
	gettimeofday(&start, NULL);	//Start time
	
	if(DeviceStatus == 0){	//If CPU does the work
		printf("GPU is not available. RGB2GRAY scale conversion done at the CPU. \n");
		RGB2GRAY(InMat, OutMat);
	}
	else{	//If GPU does the work
	
		CUDA_SAFE_CALL(hipSetDevice(MASTER));	//Setting the device
		
		//Allocate memory
		CUDA_SAFE_CALL(hipMalloc((void **)&d_InMat, ROWS*COLS*sizeof(uchar4)));
		CUDA_SAFE_CALL(hipMalloc((void **)&d_OutMat, ROWS*COLS*sizeof(uint8_t)));
		
		//Copy matrix from Host (CPU) to device (GPU)
		CUDA_SAFE_CALL(hipMemcpy(d_InMat, InMat, ROWS*COLS*sizeof(uchar4), hipMemcpyHostToDevice));
		
		/*Calculate Gridsize (which is how many blocks per grid), 
		BLOCKSIZE (which is how many threads per block)	*/
		dim3 GRIDSIZE(250,1,1);
		dim3 BLOCKSIZE(1024,1,1);
		CUDA_RGB2GRAY<<<GRIDSIZE, BLOCKSIZE>>>(d_InMat, d_OutMat);
		
		//Syncronize tasks
		
		//Copy matrix from device (GPU) to host (CPU)
		CUDA_SAFE_CALL(hipMemcpy(OutMat, d_OutMat, ROWS*COLS*sizeof(uint8_t), hipMemcpyDeviceToHost));
		
		//Free device memory
		CUDA_SAFE_CALL(hipFree(d_InMat));
		CUDA_SAFE_CALL(hipFree(d_OutMat));
	}
	
	gettimeofday(&end, NULL);	//End time
	printf("Time elpased in multiplication: %fsec. \n", ((end.tv_sec - start.tv_sec)*1000000 + end.tv_usec - start.tv_usec)/1000000.0);
	
	//Store the values in Output.txt
	if((fptr=fopen("./Output.txt","w")) == NULL){
		printf("Problem occurred while writing in Output.txt file. \n");
		return 0;
	}
	
	printf("Writing matrix in Output.txt file \n");
	for(ii=0; ii<ROWS*COLS; ii++){
		fprintf(fptr,"%d ",OutMat[ii]);
		//printf("%d ", OutMat[ii]);
		//New line
		if((ii+1)%COLS == 0){
			fprintf(fptr,"\n");
			//printf("\n");
		}
	}
	printf("Output matrix is written! \n");
	fclose(fptr);
	printf("Use MATLAB or OpenCV to check it.\n");
	
	//Free memory
	free(InMat);
	free(OutMat);
	
	return 0;

}	//End of main

